#include "hip/hip_runtime.h"
/*
 * Copyright 2018-2019 Autoware Foundation. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

//headers in local files
#include "lidar_point_pillars/scatter_cuda.h"

__global__ void scatter_kernel( int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature,
                                const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;
    int x_ind = x_coors[i_pillar];
    int y_ind = y_coors[i_pillar];
    float feature = pfe_output[i_feature*MAX_NUM_PILLARS_ + i_pillar];
    scattered_feature[i_feature*GRID_Y_SIZE*GRID_X_SIZE + y_ind * GRID_X_SIZE + x_ind] = feature;
}

ScatterCuda::ScatterCuda(const int NUM_THREADS, const int MAX_NUM_PILLARS, const int GRID_X_SIZE, const int GRID_Y_SIZE):
NUM_THREADS_(NUM_THREADS),
MAX_NUM_PILLARS_(MAX_NUM_PILLARS),
GRID_X_SIZE_(GRID_X_SIZE),
GRID_Y_SIZE_(GRID_Y_SIZE)
{
}

void ScatterCuda::doScatterCuda(const int pillar_count, int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature)
{
  scatter_kernel<<<pillar_count, NUM_THREADS_>>>(x_coors, y_coors, pfe_output, scattered_feature,
                                                MAX_NUM_PILLARS_, GRID_X_SIZE_, GRID_Y_SIZE_);
}
